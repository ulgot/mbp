/*
 * Massive Brownian Particle
 *
 * $\ddot{x} + \gamma\dot{x} = -V'(x) + a\cos(\omega t) + f + \xi(t) + \eta(t)
 *
 * see J. Spiechowicz, J. Luczka and P. Hanggi, J. Stat. Mech. (2013) P02044
 *
 */

#include <stdio.h>
#include <getopt.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define PI 3.14159265358979f

//model
__constant__ float d_amp, d_omega, d_force, d_gam, d_Dg, d_Dp, d_lambda;
__constant__ int d_comp;
float h_omega;

//simulation
float h_trans;
int h_dev, h_block, h_grid, h_spp;
long h_paths, h_periods, h_threads, h_steps, h_trigger;
__constant__ int d_spp, d_2ndorder;
__constant__ long d_paths, d_steps, d_trigger;

//output
char *h_domain;
char h_domainx, h_domainy;
float h_beginx, h_endx, h_beginy, h_endy;
int h_logx, h_logy, h_points, h_moments, h_traj, h_hist;
__constant__ char d_domainx;
__constant__ int d_points;

//vector
float *h_x, *h_v, *h_w, *h_sv, *h_sv2, *h_dx;
float *d_x, *d_v, *d_w, *d_sv, *d_sv2, *d_dx;
unsigned int *h_seeds, *d_seeds;
hiprandState *d_states;

size_t size_f, size_ui, size_p;
hiprandGenerator_t gen;

static struct option options[] = {
    {"amp", required_argument, NULL, 'a'},
    {"omega", required_argument, NULL, 'b'},
    {"force", required_argument, NULL, 'c'},
    {"gam", required_argument, NULL, 'd'},
    {"Dg", required_argument, NULL, 'e'},
    {"Dp", required_argument, NULL, 'f'},
    {"lambda", required_argument, NULL, 'g'},
    {"comp", required_argument, NULL, 'h'},
    {"dev", required_argument, NULL, 'i'},
    {"block", required_argument, NULL, 'j'},
    {"paths", required_argument, NULL, 'k'},
    {"periods", required_argument, NULL, 'l'},
    {"trans", required_argument, NULL, 'm'},
    {"spp", required_argument, NULL, 'n'},
    {"algorithm", required_argument, NULL, 'o'},
    {"mode", required_argument, NULL, 'p'},
    {"domain", required_argument, NULL, 'q'},
    {"domainx", required_argument, NULL, 'r'},
    {"domainy", required_argument, NULL, 's'},
    {"logx", required_argument, NULL, 't'},
    {"logy", required_argument, NULL, 'u'},
    {"points", required_argument, NULL, 'v'},
    {"beginx", required_argument, NULL, 'w'},
    {"endx", required_argument, NULL, 'y'},
    {"beginy", required_argument, NULL, 'z'},
    {"endy", required_argument, NULL, 'A'}
};

void usage(char **argv)
{
    printf("Usage: %s <params> \n\n", argv[0]);
    printf("Model params:\n");
    printf("    -a, --amp=FLOAT         set the AC driving amplitude 'amp' to FLOAT\n");
    printf("    -b, --omega=FLOAT       set the AC driving frequency '\\omega' to FLOAT\n");
    printf("    -c, --force=FLOAT       set the external bias 'force' to FLOAT\n");
    printf("    -d, --gam=FLOAT         set the viscosity '\\gamma' to FLOAT\n");
    printf("    -e, --Dg=FLOAT          set the Gaussian noise intensity 'Dg' to FLOAT\n");
    printf("    -f, --Dp=FLOAT          set the Poissonian noise intensity 'Dp' to FLOAT\n");
    printf("    -g, --lambda=FLOAT      set the Poissonian kicks frequency '\\lambda' to FLOAT\n\n");
    printf("    -h, --comp=INT          choose between biased and unbiased Poissonian noise. INT can be one of:\n");
    printf("                            0: biased; 1: unbiased\n");
    printf("Simulation params:\n");
    printf("    -i, --dev=INT           set the gpu device to INT\n");
    printf("    -j, --block=INT         set the gpu block size to INT\n");
    printf("    -k, --paths=LONG        set the number of paths to LONG\n");
    printf("    -l, --periods=LONG      set the number of periods to LONG\n");
    printf("    -m, --trans=FLOAT       specify fraction FLOAT of periods which stands for transients\n");
    printf("    -n, --spp=INT           specify how many integration steps should be calculated\n");
    printf("                            for a single period of the driving force\n\n");
    printf("    -o, --algorithm=STRING  sets the algorithm. STRING can be one of:\n");
    printf("                            predcorr: simplified weak order 2.0 adapted predictor-corrector\n");
    printf("                            euler: simplified weak order 1.0 regular euler-maruyama\n");
    printf("Output params:\n");
    printf("    -p, --mode=STRING       sets the output mode. STRING can be one of:\n");
    printf("                            moments: the first two moments <<v>>, <<v^2>> and diffusion coefficient\n");
    printf("                            trajectory: ensemble averaged <x>(t), <v>(t) and <x^2>(t), <v^2>(t)\n");
    printf("                            histogram: the final position x and velocity v of all paths\n");
    printf("    -q, --domain=STRING     simultaneously scan over one or two model params. STRING can be one of:\n");
    printf("                            1d: only one parameter; 2d: two parameters at once\n");
    printf("    -r, --domainx=CHAR      sets the first domain of the moments. CHAR can be one of:\n");
    printf("                            a: amp; w: omega, f: force; g: gam; D: Dg; p: Dp; l: lambda\n");
    printf("    -s, --domainy=CHAR      sets the second domain of the moments (only if --domain=2d). CHAR can be the same as above.\n");
    printf("    -t, --logx=INT          choose between linear and logarithmic scale of the domainx\n");
    printf("                            0: linear; 1: logarithmic\n");
    printf("    -u, --logy=INT          the same as above but for domainy\n");
    printf("    -v, --points=INT        set the number of samples to generate between begin and end\n");
    printf("    -w, --beginx=FLOAT      set the starting value of the domainx to FLOAT\n");
    printf("    -y, --endx=FLOAT        set the end value of the domainx to FLOAT\n");
    printf("    -z, --beginy=FLOAT      the same as --beginx, but for domainy\n");
    printf("    -A, --endy=FLOAT        the same as --endx, but for domainy\n");
    printf("\n");
}

void parse_cla(int argc, char **argv)
{
    float ftmp;
    int c, itmp;

    while( (c = getopt_long(argc, argv, "a:b:c:d:e:f:g:h:i:j:k:l:m:n:o:p:q:r:s:t:u:v:w:y:z:A", options, NULL)) != EOF) {
        switch (c) {
            case 'a':
                ftmp = atof(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_amp), &ftmp, sizeof(float));
                break;
            case 'b':
                h_omega = atof(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_omega), &h_omega, sizeof(float));
                break;
            case 'c':
                ftmp = atof(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_force), &ftmp, sizeof(float));
                break;
            case 'd':
                ftmp = atof(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_gam), &ftmp, sizeof(float));
                break;
            case 'e':
                ftmp = atof(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_Dg), &ftmp, sizeof(float));
                break;
            case 'f':
                ftmp = atof(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_Dp), &ftmp, sizeof(float));
                break;
            case 'g':
                ftmp = atof(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_lambda), &ftmp, sizeof(float));
                break;
            case 'h':
                itmp = atoi(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_comp), &itmp, sizeof(int));
                break;
            case 'i':
                itmp = atoi(optarg);
                hipSetDevice(itmp);
                break;
            case 'j':
                h_block = atoi(optarg);
                break;
            case 'k':
                h_paths = atol(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_paths), &h_paths, sizeof(long));
                break;
            case 'l':
                h_periods = atol(optarg);
                break;
            case 'm':
                h_trans = atof(optarg);
                break;
            case 'n':
                h_spp = atoi(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_spp), &h_spp, sizeof(int));
                break;
            case 'o':
                if ( !strcmp(optarg, "predcorr") )
                    itmp = 1;
                else if ( !strcmp(optarg, "euler") )
                    itmp = 0;
                hipMemcpyToSymbol(HIP_SYMBOL(d_2ndorder), &itmp, sizeof(int));
                break;
            case 'p':
                if ( !strcmp(optarg, "moments") ) {
                    h_moments = 1;
                    h_traj = 0;
                    h_hist = 0;
                } else if ( !strcmp(optarg, "trajectory") ) {
                    h_traj = 1;
                    h_hist = 0;
                    h_moments = 0;
                } else if ( !strcmp(optarg, "histogram") ) {
                    h_moments = 0;
                    h_traj = 0;
                    h_hist = 1;
                }
                break;
            case 'q':
                h_domain = optarg;
                break;
            case 'r':
                h_domainx = optarg[0]; 
                hipMemcpyToSymbol(HIP_SYMBOL(d_domainx), &h_domainx, sizeof(char));
                break;
            case 's':
                h_domainy = optarg[0];
                break;
            case 't':
                h_logx = atoi(optarg);
                break;
            case 'u':
                h_logy = atoi(optarg);
                break;
            case 'v':
                h_points = atoi(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_points), &h_points, sizeof(int));
                break;
            case 'w':
                h_beginx = atof(optarg);
                break;
            case 'y':
                h_endx = atof(optarg);
                break;
            case 'z':
                h_beginy = atof(optarg);
                break;
            case 'A':
                h_endy = atof(optarg);
                break;
        }
    }
}

__global__ void init_dev_rng(unsigned int *d_seeds, hiprandState *d_states)
{
    long idx = blockIdx.x * blockDim.x + threadIdx.x;

    hiprand_init(d_seeds[idx], idx, 0, &d_states[idx]);
}

__device__ float drift(float l_x, float l_v, float l_w, float l_gam, float l_amp, float l_force)
{
    return -l_gam*l_v - 2.0f*PI*cosf(2.0f*PI*l_x) + l_amp*cosf(l_w) + l_force;
}

__device__ float diffusion(float l_gam, float l_Dg, float l_dt, int l_2ndorder, hiprandState *l_state)
{
    if (l_Dg != 0.0f) {
        float r = hiprand_uniform(l_state);
        if (l_2ndorder) {
            if ( r <= 1.0f/6 ) {
                return -sqrtf(6.0f*l_gam*l_Dg*l_dt);
            } else if ( r > 1.0f/6 && r <= 2.0f/6 ) {
                return sqrtf(6.0f*l_gam*l_Dg*l_dt);
            } else {
                return 0.0f;
            }
        } else {
            if ( r <= 0.5f ) {
                return -sqrtf(2.0f*l_gam*l_Dg*l_dt);
            } else {
                return sqrtf(2.0f*l_gam*l_Dg*l_dt);
            }
        }
    } else {
        return 0.0f;
    }
}

__device__ float adapted_jump(int &npcd, int pcd, float l_lambda, float l_Dp, int l_comp, float l_dt, hiprandState *l_state)
{
    if (l_Dp != 0.0f) {
        float comp = sqrtf(l_Dp*l_lambda)*l_dt;
        if (pcd <= 0) {
            float ampmean = sqrtf(l_lambda/l_Dp);
           
            npcd = (int) floor( -logf( hiprand_uniform(l_state) )/l_lambda/l_dt + 0.5f );

            if (l_comp) {
                return -logf( hiprand_uniform(l_state) )/ampmean - comp;
            } else {
                return -logf( hiprand_uniform(l_state) )/ampmean;
            }
        } else {
            npcd = pcd - 1;
            if (l_comp) {
                return -comp;
            } else {
                return 0.0f;
            }
        }
    } else {
        return 0.0f;
    }
}

__device__ float regular_jump(float l_lambda, float l_Dp, int l_comp, float l_dt, hiprandState *l_state)
{
    if (l_Dp != 0.0f) {
        float mu, ampmean, comp, s;
        int i;
        unsigned int n;

        mu = l_lambda*l_dt;
        ampmean = sqrtf(l_lambda/l_Dp);
        comp = sqrtf(l_Dp*l_lambda)*l_dt;
        n = hiprand_poisson(l_state, mu);
        s = 0.0f;
            for (i = 0; i < n; i++) {
                s += -logf( hiprand_uniform(l_state) )/ampmean;
            }
        if (l_comp) s -= comp;
        return s;
    } else {
        return 0.0f;
    }
}

__device__ void predcorr(float &corrl_x, float l_x, float &corrl_v, float l_v, float &corrl_w, float l_w, int &npcd, int pcd, hiprandState *l_state, \
                         float l_amp, float l_omega, float l_force, float l_gam, float l_Dg, int l_2ndorder, float l_Dp, float l_lambda, int l_comp, float l_dt)
/* simplified weak order 2.0 adapted predictor-corrector scheme
( see E. Platen, N. Bruti-Liberati; Numerical Solution of Stochastic Differential Equations with Jumps in Finance; Springer 2010; p. 503, p. 532 )
*/
{
    float l_xt, l_xtt, l_vt, l_vtt, l_wt, l_wtt, predl_x, predl_v, predl_w;

    l_xt = l_v;
    l_vt = drift(l_x, l_v, l_w, l_gam, l_amp, l_force);
    l_wt = l_omega;

    predl_x = l_x + l_xt*l_dt;
    predl_v = l_v + l_vt*l_dt + diffusion(l_gam, l_Dg, l_dt, l_2ndorder, l_state);
    predl_w = l_w + l_wt*l_dt;

    l_xtt = predl_v;
    l_vtt = drift(predl_x, predl_v, predl_w, l_gam, l_amp, l_force);
    l_wtt = l_omega;

    predl_x = l_x + 0.5f*(l_xt + l_xtt)*l_dt;
    predl_v = l_v + 0.5f*(l_vt + l_vtt)*l_dt + diffusion(l_gam, l_Dg, l_dt, l_2ndorder, l_state);
    predl_w = l_w + 0.5f*(l_wt + l_wtt)*l_dt;

    l_xtt = predl_v;
    l_vtt = drift(predl_x, predl_v, predl_w, l_gam, l_amp, l_force);
    l_wtt = l_omega;

    corrl_x = l_x + 0.5f*(l_xt + l_xtt)*l_dt;
    corrl_v = l_v + 0.5f*(l_vt + l_vtt)*l_dt + diffusion(l_gam, l_Dg, l_dt, l_2ndorder, l_state) + adapted_jump(npcd, pcd, l_lambda, l_Dp, l_comp, l_dt, l_state);
    corrl_w = l_w + 0.5f*(l_wt + l_wtt)*l_dt;
}

__device__ void eulermaruyama(float &nl_x, float l_x, float &nl_v, float l_v, float &nl_w, float l_w, hiprandState *l_state, \
                         float l_amp, float l_omega, float l_force, float l_gam, float l_Dg, int l_2ndorder, float l_Dp, float l_lambda, int l_comp, float l_dt)
/* simplified weak order 1.0 regular euler-maruyama scheme 
( see E. Platen, N. Bruti-Liberati; Numerical Solution of Stochastic Differential Equations with Jumps in Finance; Springer 2010; p. 508, 
  C. Kim, E. Lee, P. Talkner, and P.Hanggi; Phys. Rev. E 76; 011109; 2007 ) 
*/ 
{
    float l_xt, l_vt, l_wt;

    l_vt = l_v + drift(l_x, l_v, l_w, l_gam, l_amp, l_force)*l_dt + diffusion(l_gam, l_Dg, l_dt, l_2ndorder, l_state) 
               + regular_jump(l_lambda, l_Dp, l_comp, l_dt, l_state);
    l_xt = l_x + l_v*l_dt;
    l_wt = l_w + l_omega*l_dt;

    nl_v = l_vt;
    nl_x = l_xt;
    nl_w = l_wt;
}

__device__ void fold(float &nx, float x, float y, float &nfc, float fc)
//reduce periodic variable to the base domain
{
    nx = x - floor(x/y)*y;
    nfc = fc + floor(x/y)*y;
}

__global__ void run_moments(float *d_x, float *d_v, float *d_w, float *d_sv, float *d_sv2, float *d_dx, hiprandState *d_states)
//actual moments kernel
{
    long idx = blockIdx.x * blockDim.x + threadIdx.x;
    float l_x, l_v, l_w, l_sv, l_sv2, l_dx; 
    hiprandState l_state;

    //cache path and model parameters in local variables
    l_x = d_x[idx];
    l_v = d_v[idx];
    l_w = d_w[idx];
    l_sv = d_sv[idx];
    l_sv2 = d_sv2[idx];
    l_state = d_states[idx];

    float l_amp, l_omega, l_force, l_gam, l_Dg, l_Dp, l_lambda;
    int l_comp;

    l_amp = d_amp;
    l_omega = d_omega;
    l_force = d_force;
    l_gam = d_gam;
    l_Dg = d_Dg;
    l_Dp = d_Dp;
    l_lambda = d_lambda;
    l_comp = d_comp;

    //run simulation for multiple values of the system parameters
    long ridx = (idx/d_paths) % d_points;
    l_dx = d_dx[ridx];

    switch(d_domainx) {
        case 'a':
            l_amp = l_dx;
            break;
        case 'w':
            l_omega = l_dx;
            break;
        case 'f':
            l_force = l_dx;
            break;
        case 'g':
            l_gam = l_dx;
            break;
        case 'D':
            l_Dg = l_dx;
            break;
        case 'p':
            l_Dp = l_dx;
            break;
        case 'l':
            l_lambda = l_dx;
            break;
    }

    //step size & number of steps
    float l_dt;
    long l_steps, l_trigger, i;

    l_dt = 2.0f*PI/l_omega/d_spp; 
    l_steps = d_steps;
    l_trigger = d_trigger;

    //counters for folding
    float xfc, wfc;
    
    xfc = 0.0f;
    wfc = 0.0f;

    int l_2ndorder, pcd;

    l_2ndorder = d_2ndorder;

    if (l_2ndorder) {
        //jump countdown
        pcd = (int) floor( -logf( hiprand_uniform(&l_state) )/l_lambda/l_dt + 0.5f );
    }
    
    for (i = 0; i < l_steps; i++) {

        //algorithm
        if (l_2ndorder) {
            predcorr(l_x, l_x, l_v, l_v, l_w, l_w, pcd, pcd, &l_state, l_amp, l_omega, l_force, l_gam, l_Dg, l_2ndorder, l_Dp, l_lambda, l_comp, l_dt);
        } else {
            eulermaruyama(l_x, l_x, l_v, l_v, l_w, l_w, &l_state, l_amp, l_omega, l_force, l_gam, l_Dg, l_2ndorder, l_Dp, l_lambda, l_comp, l_dt);
        }
        
        //fold path parameters
        if ( fabs(l_x) > 1.0f ) {
            fold(l_x, l_x, 1.0f, xfc, xfc);
        }

        if ( l_w > (2.0f*PI) ) {
            fold(l_w, l_w, (2.0f*PI), wfc, wfc);
        }

        if (i >= l_trigger) {
            l_sv += l_v;
            l_sv2 += l_v*l_v;
        }

    }

    //write back path parameters to the global memory
    d_x[idx] = l_x + xfc;
    d_v[idx] = l_v;
    d_w[idx] = l_w;
    d_sv[idx] = l_sv;
    d_sv2[idx] = l_sv2;
    d_states[idx] = l_state;
}

__global__ void run_traj(float *d_x, float *d_v, float *d_w, hiprandState *d_states)
//actual trajectory kernel
{
    long idx = blockIdx.x * blockDim.x + threadIdx.x;
    float l_x, l_v, l_w; 
    hiprandState l_state;

    //cache path and model parameters in local variables
    l_x = d_x[idx];
    l_v = d_v[idx];
    l_w = d_w[idx];
    l_state = d_states[idx];

    float l_amp, l_omega, l_force, l_gam, l_Dg, l_Dp, l_lambda;
    int l_comp;

    l_amp = d_amp;
    l_omega = d_omega;
    l_force = d_force;
    l_gam = d_gam;
    l_Dg = d_Dg;
    l_Dp = d_Dp;
    l_lambda = d_lambda;
    l_comp = d_comp;

    //step size & number of steps
    float l_dt;
    long l_steps, i;

    l_dt = 2.0f*PI/l_omega/d_spp; 
    l_steps = d_steps;

    //counters for folding
    float xfc, wfc;
    
    xfc = 0.0f;
    wfc = 0.0f;

    int l_2ndorder, pcd;

    l_2ndorder = d_2ndorder;

    if (l_2ndorder) {
        //jump countdown
        pcd = (int) floor( -logf( hiprand_uniform(&l_state) )/l_lambda/l_dt + 0.5f );
    }
    
    for (i = 0; i < l_steps; i++) {

        //algorithm
        if (l_2ndorder) {
            predcorr(l_x, l_x, l_v, l_v, l_w, l_w, pcd, pcd, &l_state, l_amp, l_omega, l_force, l_gam, l_Dg, l_2ndorder, l_Dp, l_lambda, l_comp, l_dt);
        } else {
            eulermaruyama(l_x, l_x, l_v, l_v, l_w, l_w, &l_state, l_amp, l_omega, l_force, l_gam, l_Dg, l_2ndorder, l_Dp, l_lambda, l_comp, l_dt);
        }
        
        //fold path parameters
        if ( fabs(l_x) > 1.0f ) {
            fold(l_x, l_x, 1.0f, xfc, xfc);
        }

        if ( l_w > (2.0f*PI) ) {
            fold(l_w, l_w, (2.0f*PI), wfc, wfc);
        }

    }

    //write back path parameters to the global memory
    d_x[idx] = l_x + xfc;
    d_v[idx] = l_v;
    d_w[idx] = l_w;
    d_states[idx] = l_state;
}

void prepare()
//prepare simulation
{
    //grid size
    h_paths = (h_paths/h_block)*h_block;
    h_threads = h_paths;

    if (h_moments) h_threads *= h_points;

    h_grid = h_threads/h_block;

    //number of steps
    if (h_traj) {
        h_steps = h_spp;
    } else {
        h_steps = h_periods*h_spp;
    }
    hipMemcpyToSymbol(HIP_SYMBOL(d_steps), &h_steps, sizeof(long));
     
    //host memory allocation
    size_f = h_threads*sizeof(float);
    size_ui = h_threads*sizeof(unsigned int);
    size_p = h_points*sizeof(float);

    h_x = (float*)malloc(size_f);
    h_v = (float*)malloc(size_f);
    h_w = (float*)malloc(size_f);
    h_seeds = (unsigned int*)malloc(size_ui);

    //create & initialize host rng
    hiprandCreateGeneratorHost(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL));

    hiprandGenerate(gen, h_seeds, h_threads);
 
    //device memory allocation
    hipMalloc((void**)&d_x, size_f);
    hipMalloc((void**)&d_v, size_f);
    hipMalloc((void**)&d_w, size_f);
    hipMalloc((void**)&d_seeds, size_ui);
    hipMalloc((void**)&d_states, h_threads*sizeof(hiprandState));

    //copy seeds from host to device
    hipMemcpy(d_seeds, h_seeds, size_ui, hipMemcpyHostToDevice);

    //initialization of device rng
    init_dev_rng<<<h_grid, h_block>>>(d_seeds, d_states);

    free(h_seeds);
    hipFree(d_seeds);

    //moments specific requirements
    if (h_moments) {
        h_trigger = h_steps*h_trans;
        hipMemcpyToSymbol(HIP_SYMBOL(d_trigger), &h_trigger, sizeof(long));

        h_sv = (float*)malloc(size_f);
        h_sv2 = (float*)malloc(size_f);
        h_dx = (float*)malloc(size_p);

        float dxtmp = h_beginx;
        float dxstep = (h_endx - h_beginx)/h_points;

        long i;
        
        //set domainx
        for (i = 0; i < h_points; i++) {
            if (h_logx) {
                h_dx[i] = pow(10.0f, dxtmp);
            } else {
                h_dx[i] = dxtmp;
            }
            dxtmp += dxstep;
        }
        
        hipMalloc((void**)&d_sv, size_f);
        hipMalloc((void**)&d_sv2, size_f);
        hipMalloc((void**)&d_dx, size_p);
    
        hipMemcpy(d_dx, h_dx, size_p, hipMemcpyHostToDevice);
    }
}

void copy_to_dev()
{
    hipMemcpy(d_x, h_x, size_f, hipMemcpyHostToDevice);
    hipMemcpy(d_v, h_v, size_f, hipMemcpyHostToDevice);
    hipMemcpy(d_w, h_w, size_f, hipMemcpyHostToDevice);
    if (h_moments) {
        hipMemcpy(d_sv, h_sv, size_f, hipMemcpyHostToDevice);
        hipMemcpy(d_sv2, h_sv2, size_f, hipMemcpyHostToDevice);
    }
}

void copy_from_dev()
{
    hipMemcpy(h_x, d_x, size_f, hipMemcpyDeviceToHost);
    hipMemcpy(h_v, d_v, size_f, hipMemcpyDeviceToHost);
    hipMemcpy(h_w, d_w, size_f, hipMemcpyDeviceToHost);
    if (h_moments) {
        hipMemcpy(h_sv, d_sv, size_f, hipMemcpyDeviceToHost);
        hipMemcpy(h_sv2, d_sv2, size_f, hipMemcpyDeviceToHost);
    }
}

void initial_conditions()
//set initial conditions for path parameters
{
    hiprandGenerateUniform(gen, h_x, h_threads); //x in (0,1]
    hiprandGenerateUniform(gen, h_v, h_threads);
    hiprandGenerateUniform(gen, h_w, h_threads);

    long i;

    for (i = 0; i < h_threads; i++) {
        h_v[i] = 4.0f*h_v[i] - 2.0f; //v in (-2,2]
        h_w[i] *= 2.0f*PI; //w in (0,2\pi]
    }

    if (h_moments) {
        memset(h_sv, 0, size_f);
        memset(h_sv2, 0, size_f);
    }
    
    copy_to_dev();
}

void moments(float *av, float *av2, float *dc)
//calculate the first two moments of <v> and diffusion coefficient
{
    float sv, sv2, sx, sx2;
    int i, j;

    hipMemcpy(h_sv, d_sv, size_f, hipMemcpyDeviceToHost);
    hipMemcpy(h_sv2, d_sv2, size_f, hipMemcpyDeviceToHost);
    hipMemcpy(h_x, d_x, size_f, hipMemcpyDeviceToHost);

    for (j = 0; j < h_points; j++) {
        sv = 0.0f;
        sv2 = 0.0f;
        sx = 0.0f;
        sx2 = 0.0f;

        for (i = 0; i < h_paths; i++) {
            sv += h_sv[j*h_paths + i];
            sv2 += h_sv2[j*h_paths + i];
            sx += h_x[j*h_paths + i];
            sx2 += h_x[j*h_paths + i]*h_x[j*h_paths + i];
        }

        av[j] = sv/(h_steps - h_trigger)/h_paths;
        av2[j] = sv2/(h_steps - h_trigger)/h_paths;
        sx /= h_paths;
        sx2 /= h_paths;
        if (h_domainx == 'w') {
            dc[j] = (sx2 - sx*sx)/(2.0f*h_periods*2.0f*PI/h_dx[j]);
        } else {
            dc[j] = (sx2 - sx*sx)/(2.0f*h_periods*2.0f*PI/h_omega);
        }
    }
}

void ensemble_average(float *h_x, float *h_v, float &sx, float &sv, float &sx2, float &sv2)
//calculate ensemble average
{
    int i;

    sx = 0.0f;
    sv = 0.0f;
    sx2 = 0.0f;
    sv2 = 0.0f;

    for (i = 0; i < h_threads; i++) {
        sx += h_x[i];
        sv += h_v[i];
        sx2 += h_x[i]*h_x[i];
        sv2 += h_v[i]*h_v[i];
    }

    sx /= h_threads;
    sv /= h_threads;
    sx2 /= h_threads;
    sv2 /= h_threads;
}

void finish()
//free memory
{

    free(h_x);
    free(h_v);
    free(h_w);
    
    hiprandDestroyGenerator(gen);
    hipFree(d_x);
    hipFree(d_v);
    hipFree(d_w);
    hipFree(d_states);
    
    if (h_moments) {
        free(h_sv);
        free(h_sv2);
        free(h_dx);

        hipFree(d_sv);
        hipFree(d_sv2);
        hipFree(d_dx);
    }
}

int main(int argc, char **argv)
{
    parse_cla(argc, argv);
    if (!h_moments && !h_traj && !h_hist) {
        usage(argv);
        return -1;
    }

    prepare();
    
    initial_conditions();
    
    //asymptotic long time average velocity <<v>>, <<v^2>> and diffusion coefficient
    if (h_moments) {
        float *av, *av2, *dc;
        int i;

        av = (float*)malloc(size_p);
        av2 = (float*)malloc(size_p);
        dc = (float*)malloc(size_p);

        if ( !strcmp(h_domain, "1d") ) {
            run_moments<<<h_grid, h_block>>>(d_x, d_v, d_w, d_sv, d_sv2, d_dx, d_states);
            moments(av, av2, dc);

            printf("#%c <<v>> <<v^2>> D_x\n", h_domainx);
            for (i = 0; i < h_points; i++) {
                printf("%e %e %e %e\n", h_dx[i], av[i], av2[i], dc[i]);
            }

        } else {
            float h_dy, dytmp, dystep;
            int j;
            
            dytmp = h_beginy;
            dystep = (h_endy - h_beginy)/h_points;
            
            printf("#%c %c <<v>> <<v^2>> D_x\n", h_domainx, h_domainy);
            
            for (i = 0; i < h_points; i++) {
                if (h_logy) {
                    h_dy = pow(10.0f, dytmp);
                } else {
                    h_dy = dytmp;
                }

                switch(h_domainy) {
                    case 'a':
                        hipMemcpyToSymbol(HIP_SYMBOL(d_amp), &h_dy, sizeof(float));
                        break;
                    case 'w':
                        h_omega = h_dy;
                        hipMemcpyToSymbol(HIP_SYMBOL(d_omega), &h_omega, sizeof(float));
                        break;
                    case 'f':
                        hipMemcpyToSymbol(HIP_SYMBOL(d_force), &h_dy, sizeof(float));
                        break;
                    case 'g':
                        hipMemcpyToSymbol(HIP_SYMBOL(d_gam), &h_dy, sizeof(float));
                        break;
                    case 'D':
                        hipMemcpyToSymbol(HIP_SYMBOL(d_Dg), &h_dy, sizeof(float));
                        break;
                    case 'p':
                        hipMemcpyToSymbol(HIP_SYMBOL(d_Dp), &h_dy, sizeof(float));
                        break;
                    case 'l':
                        hipMemcpyToSymbol(HIP_SYMBOL(d_lambda), &h_dy, sizeof(float));
                        break;
                }

                run_moments<<<h_grid, h_block>>>(d_x, d_v, d_w, d_sv, d_sv2, d_dx, d_states);
                moments(av, av2, dc);
                
                for (j = 0; j < h_points; j++) {
                    printf("%e %e %e %e %e\n", h_dx[j], h_dy, av[j], av2[j], dc[j]);
                }

                //blank line for plotting purposes
                printf("\n");

                initial_conditions();

                dytmp += dystep;
            }
        }

        free(av);
        free(av2);
        free(dc);
    }

    //ensemble averaged trajectory <x>(t), <v>(t) and <x^2>(t), <v^2>(t)
    if (h_traj) {
        float t, sx, sv, sx2, sv2;
        int i;

        printf("#t <x> <v> <x^2> <v^2>\n");

        for (i = 0; i < h_periods; i++) {
            run_traj<<<h_grid, h_block>>>(d_x, d_v, d_w, d_states);
            copy_from_dev();
            t = i*2.0f*PI/h_omega;
            ensemble_average(h_x, h_v, sx, sv, sx2, sv2);
            printf("%e %e %e %e %e\n", t, sx, sv, sx2, sv2);
        }
    }

    //the final position x and velocity v of all paths
    if (h_hist) {
        int i;

        run_traj<<<h_grid, h_block>>>(d_x, d_v, d_w, d_states);
        copy_from_dev();

        printf("#x v\n");
        
        for (i = 0; i < h_threads; i++) {
            printf("%e %e\n", h_x[i], h_v[i]); 
        }
    }

    finish();

    return 0;
}
